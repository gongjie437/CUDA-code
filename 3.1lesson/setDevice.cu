
#include <hip/hip_runtime.h>
#include <stdio.h>


int main(void)
{
    // 检测计算机GPU数量
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);
//if you execute export CUDA_VISIBLE_DEVICES=0,1,2 in the same bash windows as this exeuctable, this will get GPU device count as 3 
    if (error != hipSuccess || iDeviceCount == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    else
    {
        printf("The count of GPUs is %d.\n", iDeviceCount);
    }
    
    // 设置执行
    int iDev = 0;
    error = hipSetDevice(iDev); 
    if (error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing.\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing.\n");
    }

    return 0;
}

