
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;

    const int id = threadIdx.x + blockIdx.x * blockDim.x; 
    printf("Hello World from block %d and thread %d, global id %d\n", bid, tid, id);
}


int main(void)
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();

    return 0;
}


/*
The deprecated cudaThreadSynchronize() function in CUDA has been replaced by the recommended cudaDeviceSynchronize() function. Let’s delve into the details:

cudaThreadSynchronize():
This function waits for all previous asynchronous operations (such as kernel calls and asynchronous memory copies) to complete.
It halts execution in the CPU/host thread until the GPU finishes processing all previously requested CUDA tasks.
However, it is deprecated, meaning it still works for now but is not recommended for future use.
cudaDeviceSynchronize():
Similar to cudaThreadSynchronize(), it also waits for all previous CUDA tasks to complete.
It is the preferred replacement for cudaThreadSynchronize().
Use cudaDeviceSynchronize() to ensure synchronization between the CPU and GPU.
In summary, if you were using cudaThreadSynchronize(), consider switching to cudaDeviceSynchronize() for better compatibility and future-proofing your code1234. 🚀


*/
